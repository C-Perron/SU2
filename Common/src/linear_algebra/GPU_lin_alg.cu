#include "hip/hip_runtime.h"
#include<hip/hip_runtime.h>
#include "../../include/linear_algebra/CSysMatrix.inl"
#include "../../include/linear_algebra/CSysMatrix.hpp"
#include "../../include/geometry/CGeometry.hpp"

template<class ScalarType>
void CSysMatrix<ScalarType>::GPUMatrixStructMemAlloc()
{
    
}

__global__ void GPUMatrixVectorProductAdd(su2mixedfloat* matrix, double* vec, double* prod, unsigned long* d_row_ptr, unsigned long* d_col_ind, unsigned long nPointDomain, unsigned long nVar, unsigned long nEqn)
{
   int i = blockIdx.x * blockDim.x + threadIdx.x;
   int j = threadIdx.y;
   int k = threadIdx.z;

   if(i<nPointDomain)
   {
      for(int index = d_row_ptr[i]; index<d_row_ptr[i+1]; index++)
      {
         int matrix_index = index * nVar * nEqn;
         int vec_index = d_col_ind[index] * nEqn;
         int prod_index = i * nVar;
      
         prod[prod_index + k] += matrix[ matrix_index + (i * nEqn + j)] * vec[vec_index + j];

      }
   }
}


template<class ScalarType>
void CSysMatrix<ScalarType>::GPUMatrixVectorProduct(const CSysVector<ScalarType>& vec, CSysVector<ScalarType>& prod,
                                                 CGeometry* geometry, const CConfig* config) const
                                                 {


  double* d_vec;
  double* d_prod;

  unsigned long nPointDomain = geometry->GetnPointDomain();
  unsigned long nDim = geometry->GetnDim();
  unsigned long nVar = nDim + 2 ;
  unsigned long nEqn = nVar;

  unsigned long mat_size = nnz*nVar*nEqn;
  unsigned long vec_size = nPointDomain*nVar;

  hipMalloc((void**)(&d_vec), (sizeof(&vec[0])*vec_size));
  hipMalloc((void**)(&d_prod), (sizeof(&prod[0])*vec_size));

  hipError_t code = hipGetLastError();
        if(code != hipSuccess)
        {
            std::cerr << code << " Error Code " << std::endl;
        }

  hipMemcpy((void*)(d_matrix), (void*)&matrix[0], (sizeof(&matrix[0])*mat_size), hipMemcpyHostToDevice);
  hipMemcpy((void*)(d_vec), (void*)&vec[0], (sizeof(&vec[0])*vec_size), hipMemcpyHostToDevice);
  hipMemcpy((void*)(d_prod), (void*)&prod[0], (sizeof(&prod[0])*vec_size), hipMemcpyHostToDevice);

  code = hipGetLastError();
        if(code != hipSuccess)
        {
            std::cerr << code << " Error Code " << std::endl;
        }

  long xDim = floor(512.0/(nVar*nEqn));
  dim3 blockDim(xDim, nEqn, nVar);
  dim3 gridDim(ceil(nPointDomain/xDim), 1, 1);

  GPUMatrixVectorProductAdd<<<gridDim, blockDim>>>(d_matrix, d_vec, d_prod, d_row_ptr, d_col_ind, nPointDomain, nVar, nEqn);

  code = hipGetLastError();
        if(code != hipSuccess)
        {
            std::cerr << code << " Error Code " << std::endl;
        }

  hipMemcpy((void*)(&prod[0]), (void*)d_prod, (sizeof(&prod[0])*vec_size), hipMemcpyDeviceToHost);

  
  hipFree(d_vec);
  hipFree(d_prod);

}

template class CSysMatrix<su2mixedfloat>;