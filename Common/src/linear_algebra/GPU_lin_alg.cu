#include "hip/hip_runtime.h"
#include<hip/hip_runtime.h>
#include "../../include/linear_algebra/CSysMatrix.inl"
#include "../../include/linear_algebra/CSysMatrix.hpp"
#include "../../include/geometry/CGeometry.hpp"

__global__ void GPUMatrixVectorProductAdd(su2mixedfloat* matrix, double* vec, double* prod, unsigned long* d_row_ptr, unsigned long* d_col_ind, unsigned long nPointDomain, unsigned long nVar, unsigned long nEqn)
{
   int i = blockIdx.x * blockDim.x + threadIdx.x;
   int j = threadIdx.y;
   int k = threadIdx.z;

   if(i<nPointDomain)
   {
      for(int index = d_row_ptr[i]; index<d_row_ptr[i+1]; index++)
      {
         int matrix_index = index * nVar * nEqn;
         int vec_index = d_col_ind[index] * nEqn;
         int prod_index = i * nVar;
      
         prod[prod_index + k] += matrix[ matrix_index + (i * nEqn + j)] * vec[vec_index + j];

      }
   }
}



template<class ScalarType>
void CSysMatrix<ScalarType>::GPUMatrixVectorProduct(const CSysVector<ScalarType>& vec, CSysVector<ScalarType>& prod,
                                                 CGeometry* geometry, const CConfig* config) const
                                                 {

  unsigned long* d_row_ptr; /*!< \brief Device Pointers to the first element in each row. */
  unsigned long* d_col_ind; /*!< \brief Device Column index for each of the elements in val(). */

  su2mixedfloat* d_matrix;
  double* d_vec;
  double* d_prod;

  unsigned long nPointDomain = geometry->GetnPointDomain();
  unsigned long nDim = geometry->GetnDim();
  unsigned long nVar = nDim + 2 ;
  unsigned long nEqn = nVar;

  unsigned long mat_size = nnz*nVar*nEqn;
  unsigned long vec_size = nPointDomain*nVar;

  hipMalloc((void**)(&d_row_ptr), (sizeof(row_ptr)*(nPointDomain+1.0)));
  hipMalloc((void**)(&d_col_ind), (sizeof(col_ind)*nnz));
  hipMalloc((void**)(&d_matrix), (sizeof(&matrix[0])*mat_size));
  hipMalloc((void**)(&d_vec), (sizeof(&vec[0])*vec_size));
  hipMalloc((void**)(&d_prod), (sizeof(&prod[0])*vec_size));

  hipError_t code1 = hipGetLastError();
        if(code1 != hipSuccess)
        {
            std::cerr << code1 << " Error Code " << std::endl;
        }

  hipMemcpy((void*)(d_row_ptr), (void*)row_ptr, (sizeof(row_ptr)*(nPointDomain+1.0)), hipMemcpyHostToDevice);
  hipMemcpy((void*)(d_col_ind), (void*)col_ind, (sizeof(col_ind))*nnz, hipMemcpyHostToDevice);
  hipMemcpy((void*)(d_matrix), (void*)&matrix[0], (sizeof(&matrix[0])*mat_size), hipMemcpyHostToDevice);
  hipMemcpy((void*)(d_vec), (void*)&vec[0], (sizeof(&vec[0])*vec_size), hipMemcpyHostToDevice);
  hipMemcpy((void*)(d_prod), (void*)&prod[0], (sizeof(&prod[0])*vec_size), hipMemcpyHostToDevice);

  hipError_t code2 = hipGetLastError();
        if(code2 != hipSuccess)
        {
            std::cerr << code2 << " Error Code " << std::endl;
        }

  long xDim = floor(512.0/(nVar*nEqn));
  dim3 blockDim(xDim, nEqn, nVar);
  dim3 gridDim(ceil(nPointDomain/xDim), 1, 1);

  GPUMatrixVectorProductAdd<<<gridDim, blockDim>>>(d_matrix, d_vec, d_prod, d_row_ptr, d_col_ind, nPointDomain, nVar, nEqn);

  hipError_t code3 = hipGetLastError();
        if(code3 != hipSuccess)
        {
            std::cerr << code3 << " Error Code " << std::endl;
        }

  hipMemcpy((void*)(&prod[0]), (void*)d_prod, (sizeof(&prod[0])*vec_size), hipMemcpyDeviceToHost);

  hipFree(d_col_ind);
  hipFree(d_row_ptr);
  hipFree(d_vec);
  hipFree(d_prod);
  hipFree(d_matrix);

}

template class CSysMatrix<su2mixedfloat>;